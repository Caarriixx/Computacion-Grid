/*
 * dgemm_gpu_shared.cu
 *
 * compile with: make dgemm_gpu_shared
 *    -> nvcc -O3 -arch=sm_20 -o dgemm_gpu_shared dgemm_gpu_shared.cu -lcudart
 *
 * Matrices are stored as array in row-major order: 
 * A[row][col] = A[row * N + col]
 *
 * Use shared memory to speed up the matrix multiplication. We can reuse
 * the memory if we load a block of the matrix and have a thread block 
 * calculate a sub matrix.
 */


#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// Thread block size: BLOCK_SIZE * BLOCK_SIZE
#define BLOCK_SIZE 16

// Declaration of helper functions (see bottom of file for details)
void checkError (const char* action);
float getGflops (int, float);

/*
 *  Matrix multiplication kernel called by matrixMulOnDevice() 
 */
__global__ void dgemm_gpu_shared(double* a, double* b, double* c, int n){
    
    // TODO: Allocate shared memory for the two blocks aSub and bSub.
    //       Use two-dimensional matrices of size BLOCK_SIZE * BLOCK_SIZE 
    __shared__ double aSub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double bSub[BLOCK_SIZE][BLOCK_SIZE];

    
    // TODO: Calculate global thread index 
    int idxX = blockIdx.x * blockDim.x + threadIdx.x;
    int idxY = blockIdx.y * blockDim.y + threadIdx.y;

    
    // For the matrix multiplication, we need to multiply all the elements of 
    // the idxYth row of a with all the elements of the idXth column of b and 
    // sum up the results.
    double sum = 0;

    // TODO: Calculate global offset of upper left corner of thread block.
    int blockaY = blockIdx.y * BLOCK_SIZE;
    int blockbX = blockIdx.x * BLOCK_SIZE;


    for (int block = 0; block < gridDim.x; ++block){
        // Get the two sub matrices
        int blockaX = block * (BLOCK_SIZE);
        int blockbY = block * (BLOCK_SIZE);
        if (((blockaY + threadIdx.y) < n) && (blockaX + threadIdx.x) < n) {
          // TODO: Copy block into shared memory
          aSub[threadIdx.y][threadIdx.x] = a[(blockaY + threadIdx.y) * n + (blockaX + threadIdx.x)];
        } else {
          aSub[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (((blockbY + threadIdx.y) < n) && (blockbX + threadIdx.x) < n) {
          bSub[threadIdx.y][threadIdx.x] = b[(blockbY + threadIdx.y) * n + (blockbX + threadIdx.x)];
        } else {
          bSub[threadIdx.y][threadIdx.x] = 0.0; 
        }
	
		// TODO: Synchronize threads to make sure all threads are done copying
    __syncthreads();
    
        if ((idxX < n) && (idxY < n))
        {
            for (int i=0; i < blockDim.x; ++i){ //assumes that we use square blocks
                sum += aSub[threadIdx.y][i] * bSub[i][threadIdx.x];
            }
        }

		// TODO: Synchronize threads to make sure all threads are done with the data
    __syncthreads();

    }
    if ((idxX < n) && (idxY < n)){    
        c[idxY * n + idxX] = sum;
    }
}



/*
 *  Matrix multiplication host function called by main() 
 */

void matrixMulOnDevice(double* a, double* b, double* c, int n)
{
    int size = n * n * sizeof(double);
    int xGrid, yGrid;
    double *d_a, *d_b, *d_c;
    float time;

    // Define events for timing
    hipEvent_t start, stop;
  
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // TODO
    // Allocate memory for d_a, d_b and d_c on device

    hipMalloc((void**)&d_a, size);
    checkError("cudaMalloc: d_a");

    hipMalloc((void**)&d_b, size);
    checkError("cudaMalloc: d_b");  

    hipMalloc((void**)&d_c, size);
    checkError("cudaMalloc: d_c");
    
    
    // First calculate grid size by dividing n by BLOCK_SIZE = 16
    xGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
    yGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
    dim3 gridDim(xGrid, yGrid);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    printf("Grid: %d, %d; block:%d, %d\n", xGrid , yGrid , BLOCK_SIZE, BLOCK_SIZE);
    
    // Invoke kernel and measure execution time 
    hipEventRecord( start, 0 );
    
    // TODO
    // Copy data for a and b from host to device

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    checkError("copying data of A from host to device");

    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    checkError("copying data of B from host to device");
  
    // TODO: Call the kernel 
    
    dgemm_gpu_shared<<<gridDim, blockDim>>>(d_a, d_b, d_c, n);
    checkError("launching kernel dgemm_gpu_shared");

    // TODO
    // Read restults from device memory to C 
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    checkError("hipMemcpyDeviceToHost");
    
    // Get elapsed time for kernel execution
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    checkError("executing Kernel");
    hipEventElapsedTime( &time, start, stop );
    hipEventDestroy( start );
    hipEventDestroy( stop );

    printf ("\nKernel Execution Time: %f ms (dim C: %d * %d)", time, n, n);
    printf ("\nThis corresponds to: %4.4f GFLOPS", getGflops(n, time));
  
    
    // Free device memory 
    hipFree(d_a);
    checkError("Freeing d_a");
    hipFree(d_b);
    checkError("Freeing d_b");
    hipFree(d_c);
    checkError("Freeing d_c");

}

int main(int argc, char** argv)
{
  int n = 1024;
  double *a, *b, *c;
  int row, col;
  double absError, maxAbsError = 0.0, sumAbsError = 0.0;
 
  if (argc > 1) {
    n = atoi(argv[1]);
  }

    
  // show banner
  printf ("\n\n     Matrix-Multiplication \n");
  printf (    "     ==========================================\n");
  printf (  "\n     Simple DGEMM implemantation on GPU");  

  // echo device data
  int idevice = 0;
  hipSetDevice(idevice);
  hipDeviceProp_t dprops;
  hipGetDeviceProperties( &dprops, idevice );
  printf ("\n     Device name = %s, with compute capability %d.%d \n", 
	  dprops.name, dprops.major, dprops.minor);
  printf (  "\n     Matrix size %d x %d", n, n);
  
    
  // Allocate memory for matrices on host
  assert ( a = (double*) malloc (n * n * sizeof(double)) );
  assert ( b = (double*) malloc (n * n * sizeof(double)) );
  assert ( c = (double*) malloc (n * n * sizeof(double)) );
    
  // Init matrices A and B: A = E so result will be B
#pragma omp parallel for private(row, col)
  for (row = 0; row < n; ++row){
      for (col = 0; col < n; col++){
	a[row * n + col] = (row == col) ? 1.0 : 0.0;
	b[row * n + col] = row * n + col;
      }
    }

    // do matrix multiplication on device
    matrixMulOnDevice(a, b, c, n);
     
    // Compare results
    for ( row = 0; row < n; ++row){
      for ( col = 0; col < n; ++col) {
	
	absError = fabs ( c[row * n + col] - b[row * n + col]);
	sumAbsError += absError;
	
	if (absError > maxAbsError)
	  maxAbsError = absError;
      }
    }
    // Free memory on host
    free (a);
    free (b);
    free (c);
  
    printf ("\nmaxAbsError: %4.4f, sumAbsError: %4.4f", maxAbsError, sumAbsError);
    if (maxAbsError < 2.0e-5)
      printf ("\n\nProgram terminated SUCCESSFULLY.\n\n");

    return 0;
}

/*
 *  Some helper functions
 */

// get compute performance
float getGflops (int n, float time) {

	float gf = (2.0e-6 * n * n* n / time);

	return gf;
}

// Simple error checking function for CUDA actions

void checkError (const char* action) {
  
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_FAILURE);
  }
}
