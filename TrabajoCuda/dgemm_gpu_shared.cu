
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define N 512
#define TILE_WIDTH 16

__global__ void matrixMulShared(float* A, float* B, float* C, int width) {
    // Allocate shared memory for the two blocks aSub and bSub.
    __shared__ float aSub[TILE_WIDTH][TILE_WIDTH];
    __shared__ float bSub[TILE_WIDTH][TILE_WIDTH];

    // Calculate global thread index 
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate global offset of upper left corner of thread block.
    int Row = blockIdx.y * TILE_WIDTH + ty;
    int Col = blockIdx.x * TILE_WIDTH + tx;

    float sum = 0.0f;
    for (int t = 0; t < width / TILE_WIDTH; ++t) {
        // Copy block into shared memory
        aSub[ty][tx] = A[Row * width + (t * TILE_WIDTH + tx)];
        bSub[ty][tx] = B[(t * TILE_WIDTH + ty) * width + Col];

        // Synchronize threads to make sure all threads are done copying
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i)
            sum += aSub[ty][i] * bSub[i][tx];

        // Synchronize threads to make sure all threads are done with the data
        __syncthreads();
    }

    if (Row < width && Col < width)
        C[Row * width + Col] = sum;
}

int main() {
    int size = N * N * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(N / TILE_WIDTH, N / TILE_WIDTH);

    // Call the kernel 
    matrixMulShared<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Sample result C[0]: %f\n", h_C[0]);

    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
