#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 1024  // tamaño del vector
#define RADIUS 1  // radio del stencil

__global__ void stencil_1d_not_shared(int *in, int *out, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= RADIUS && i < n - RADIUS) {
        out[i] = (in[i - 1] + in[i] + in[i + 1]) / 3;
    }
}

__global__ void stencil_1d_shared(int *in, int *out, int n) {
    extern __shared__ int temp[];
    int tid = threadIdx.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // cargar a memoria compartida
    if (i < n) {
        temp[tid + RADIUS] = in[i];
        if (tid == 0 && i > 0)
            temp[tid] = in[i - 1];
        if (tid == blockDim.x - 1 && i < n - 1)
            temp[tid + 2 * RADIUS] = in[i + 1];
    }

    __syncthreads();

    if (i >= RADIUS && i < n - RADIUS) {
        out[i] = (temp[tid] + temp[tid + 1] + temp[tid + 2]) / 3;
    }
}

void init_vector(int *v, int n) {
    for (int i = 0; i < n; i++) {
        v[i] = rand() % 100;
    }
}

void print_vector(const char *name, int *v, int n) {
    printf("%s:\n", name);
    for (int i = 0; i < n; i++) {
        printf("%d ", v[i]);
    }
    printf("\n");
}

int main(int argc, char *argv[]) {
    int blockSize = 256;  // valor por defecto

    if (argc > 1) {
        blockSize = atoi(argv[1]);
    }

    int *in, *out1, *out2;
    int *d_in, *d_out1, *d_out2;

    size_t size = N * sizeof(int);
    in = (int *)malloc(size);
    out1 = (int *)malloc(size);
    out2 = (int *)malloc(size);

    hipMalloc(&d_in, size);
    hipMalloc(&d_out1, size);
    hipMalloc(&d_out2, size);

    srand(time(NULL));
    init_vector(in, N);

    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    int numBlocks = (N + blockSize - 1) / blockSize;

    // tiempo no_shared
    hipEvent_t start1, stop1;
    hipEventCreate(&start1); hipEventCreate(&stop1);
    hipEventRecord(start1);
    stencil_1d_not_shared<<<numBlocks, blockSize>>>(d_in, d_out1, N);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float time1;
    hipEventElapsedTime(&time1, start1, stop1);

    // tiempo shared
    hipEvent_t start2, stop2;
    hipEventCreate(&start2); hipEventCreate(&stop2);
    hipEventRecord(start2);
    stencil_1d_shared<<<numBlocks, blockSize, (blockSize + 2 * RADIUS) * sizeof(int)>>>(d_in, d_out2, N);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    float time2;
    hipEventElapsedTime(&time2, start2, stop2);

    hipMemcpy(out1, d_out1, size, hipMemcpyDeviceToHost);
    hipMemcpy(out2, d_out2, size, hipMemcpyDeviceToHost);

    // Imprimir resultados
    print_vector("Input", in, 10);
    print_vector("Stencil not shared", out1, 10);
    print_vector("Stencil shared", out2, 10);

    printf("Tiempo sin shared memory: %.3f ms\n", time1);
    printf("Tiempo con shared memory: %.3f ms\n", time2);

    free(in); free(out1); free(out2);
    hipFree(d_in); hipFree(d_out1); hipFree(d_out2);

    return 0;
}
