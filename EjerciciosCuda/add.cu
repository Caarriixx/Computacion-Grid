
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main()
{
    int a, b, c;             // host copies of a, b, c
    int *d_a, *d_b, *d_c;    // device copies of a, b, c

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, sizeof(int));
    hipMalloc((void **)&d_b, sizeof(int));
    hipMalloc((void **)&d_c, sizeof(int));

    // Set values
    a = 2;
    b = 7;

    // Copy inputs to device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    printf("Resultado: %d\n", c);

    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
